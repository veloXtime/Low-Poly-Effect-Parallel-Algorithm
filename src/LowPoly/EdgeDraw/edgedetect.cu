#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "edgedraw.h"

__constant__ int SOBEL_X[3][3] = {{-1, 0, 1}, {-2, 0, 2}, {-1, 0, 1}};
__constant__ int SOBEL_Y[3][3] = {{-1, -2, -1}, {0, 0, 0}, {1, 2, 1}};
__constant__ unsigned char SUPPRESS_THRESHOLD = GRADIENT_THRESH;
__constant__ int ANCHORS_THRESHOLD = ANCHOR_THRESH;
__constant__ int SMALL_BLOCK_LENGTH = smallBlockLength;

__device__ void drawEdgesFromAnchorKernel(
    int x, int y, unsigned char *d_gradient, float *d_direction,
    unsigned char *d_edge, const bool horizontal, int width, int height);

__global__ void colorToGrayKernel(unsigned char *image,
                                  unsigned char *grayImage, int width,
                                  int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int pixels = width * height;

    for (int px = x * SMALL_BLOCK_LENGTH;
         px < width && px < (x + 1) * SMALL_BLOCK_LENGTH; ++px)
        for (int py = y * SMALL_BLOCK_LENGTH;
             py < height && py < (y + 1) * SMALL_BLOCK_LENGTH; ++py) {
            int idx = py * width + px;
            unsigned char r = image[idx];
            unsigned char g = image[idx + pixels];
            unsigned char b = image[idx + 2 * pixels];
            unsigned char grayValue = 0.299f * r + 0.587f * g + 0.114f * b;
            grayImage[idx] = grayValue;
        }
}

__global__ void gradientCalculationKernel(unsigned char *grayImage,
                                          unsigned char *gradient,
                                          float *direction, int width,
                                          int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    for (int px = x * SMALL_BLOCK_LENGTH;
         px > 0 && px < width - 1 && px < (x + 1) * SMALL_BLOCK_LENGTH; ++px)
        for (int py = y * SMALL_BLOCK_LENGTH;
             py > 0 && py < height - 1 && py < (y + 1) * SMALL_BLOCK_LENGTH;
             ++py) {
            int gradientX = 0, gradientY = 0;

            // Apply the Sobel filter
            for (int i = -1; i <= 1; i++) {
                for (int j = -1; j <= 1; j++) {
                    int pixel = grayImage[(py + j) * width + (px + i)];
                    gradientX += SOBEL_X[i + 1][j + 1] * pixel;
                    gradientY += SOBEL_Y[i + 1][j + 1] * pixel;
                }
            }

            int idx = py * width + px;
            gradient[idx] =
                sqrtf(gradientX * gradientX + gradientY * gradientY);
            direction[idx] = atan2f(gradientY, gradientX) * 180 / M_PI;
        }
}

void gradientInGrayGPU(CImg &image, CImg &gradient, CImgFloat &direction) {
    int width = image.width(), height = image.height();

    // Flatten the image data for CUDA
    unsigned char *d_image, *d_grayImage, *d_gradient;
    float *d_direction;
    size_t imageSize = width * height * 3 * sizeof(unsigned char);
    size_t grayImageSize = width * height * sizeof(unsigned char);
    size_t directionSize = width * height * sizeof(float);

    hipMalloc(&d_image, imageSize);
    hipMalloc(&d_grayImage, grayImageSize);
    hipMalloc(&d_gradient, grayImageSize);
    hipMalloc(&d_direction, directionSize);

    hipMemcpy(d_image, image.data(), imageSize, hipMemcpyHostToDevice);
    hipMemset(d_gradient, 0, grayImageSize);
    hipMemset(d_direction, 0, directionSize);

    dim3 blockSize(16, 16);
    dim3 gridSize(
        ((width + smallBlockLength - 1) / smallBlockLength + blockSize.x - 1) /
            blockSize.x,
        ((height + smallBlockLength - 1) / smallBlockLength + blockSize.y - 1) /
            blockSize.y);

    colorToGrayKernel<<<gridSize, blockSize>>>(d_image, d_grayImage, width,
                                               height);
    gradientCalculationKernel<<<gridSize, blockSize>>>(
        d_grayImage, d_gradient, d_direction, width, height);

    // Copy results back to host
    hipMemcpy(gradient.data(), d_gradient, grayImageSize,
               hipMemcpyDeviceToHost);
    hipMemcpy(direction.data(), d_direction, directionSize,
               hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_image);
    hipFree(d_grayImage);
    hipFree(d_gradient);
    hipFree(d_direction);
}

__global__ void suppressWeakGradientsKernel(unsigned char *gradient, int width,
                                            int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    for (int px = x * SMALL_BLOCK_LENGTH;
         px < width && px < (x + 1) * SMALL_BLOCK_LENGTH; ++px)
        for (int py = y * SMALL_BLOCK_LENGTH;
             py < height && py < (y + 1) * SMALL_BLOCK_LENGTH; ++py) {
            int idx =
                py * width +
                px;  // Index into the 1D array representation of the image
            if (gradient[idx] <= SUPPRESS_THRESHOLD) {
                gradient[idx] = 0;
            }
        }
}

void suppressWeakGradientsGPU(CImg &gradient) {
    int width = gradient.width(), height = gradient.height();
    size_t numPixels = width * height;
    unsigned char *d_gradient;

    // Allocate GPU memory
    hipMalloc(&d_gradient, numPixels * sizeof(unsigned char));

    // Copy data from host to device
    hipMemcpy(d_gradient, gradient.data(), numPixels * sizeof(unsigned char),
               hipMemcpyHostToDevice);

    // Define block and grid sizes
    dim3 blockSize(16, 16);
    dim3 gridSize(
        ((width + smallBlockLength - 1) / smallBlockLength + blockSize.x - 1) /
            blockSize.x,
        ((height + smallBlockLength - 1) / smallBlockLength + blockSize.y - 1) /
            blockSize.y);

    // Launch the kernel
    suppressWeakGradientsKernel<<<gridSize, blockSize>>>(d_gradient, width,
                                                         height);

    // Copy the modified data back to the host
    hipMemcpy(gradient.data(), d_gradient, numPixels * sizeof(unsigned char),
               hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_gradient);
}

__device__ bool isHorizontalCuda(float angle) {
    if ((angle < 45 && angle >= -45) || angle >= 136 || angle < -135) {
        return true;  // horizontal
    } else {
        return false;
    }
}

__global__ void determineAnchorsKernel(unsigned char *gradient,
                                       float *direction, bool *anchor,
                                       int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x > 0 && x < width - 1 && y > 0 && y < height - 1 && x % 2 == 0 &&
        y % 2 == 0) {
        float angle = direction[y * width + x];
        int magnitude = gradient[y * width + x];
        int mag1 = 0, mag2 = 0;

        if (isHorizontalCuda(angle)) {
            mag1 = gradient[(y - 1) * width + x];
            mag2 = gradient[(y + 1) * width + x];
        } else {
            mag1 = gradient[y * width + (x - 1)];
            mag2 = gradient[y * width + (x + 1)];
        }

        bool is_anchor = (magnitude - mag1 >= ANCHORS_THRESHOLD) &&
                         (magnitude - mag2 >= ANCHORS_THRESHOLD);
        anchor[y * width + x] = is_anchor;
    }
}

void determineAnchorsGPU(const CImg &gradient, const CImgFloat &direction,
                         CImgBool &anchor) {
    int width = gradient.width();
    int height = gradient.height();
    size_t numPixels = width * height;

    // Device memory pointers
    unsigned char *d_gradient;
    float *d_direction;
    bool *d_anchor;

    // Allocate device memory
    hipMalloc(&d_gradient, numPixels * sizeof(unsigned char));
    hipMalloc(&d_direction, numPixels * sizeof(float));
    hipMalloc(&d_anchor, numPixels * sizeof(bool));

    // Copy data to device
    hipMemcpy(d_gradient, gradient.data(), numPixels * sizeof(unsigned char),
               hipMemcpyHostToDevice);
    hipMemcpy(d_direction, direction.data(), numPixels * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemset(d_anchor, 0, numPixels * sizeof(bool));

    // Kernel launch parameters
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                  (height + blockSize.y - 1) / blockSize.y);

    // Launch kernel
    determineAnchorsKernel<<<gridSize, blockSize>>>(d_gradient, d_direction,
                                                    d_anchor, width, height);

    // Copy results back to host
    hipMemcpy(anchor.data(), d_anchor, numPixels * sizeof(bool),
               hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_gradient);
    hipFree(d_direction);
    hipFree(d_anchor);
}

__device__ bool validCuda(int x, int y, int width, int height) {
    return x > 0 && y > 0 && x < width - 1 && y < height - 1;
}

__device__ void drawHorizontalEdgeFromAnchorKernel(int x, int y,
                                                   unsigned char *d_gradient,
                                                   float *d_direction,
                                                   unsigned char *d_edge,
                                                   int width, int height) {
    if (!validCuda(x, y, width, height) || d_edge[y * width + x]) return;

    int curr_x = x;
    int curr_y = y;
    d_edge[y * width + x] = 0;
    while (validCuda(curr_x, curr_y, width, height) &&
           d_gradient[curr_y * width + curr_x] > 0 &&
           !d_edge[curr_y * width + curr_x] &&
           isHorizontalCuda(d_direction[curr_y * width + curr_x])) {
        d_edge[curr_y * width + curr_x] = 255;
        unsigned char leftUp = d_gradient[(curr_y - 1) * width + curr_x - 1];
        unsigned char left = d_gradient[curr_y * width + curr_x - 1];
        unsigned char leftDown = d_gradient[(curr_y + 1) * width + curr_x - 1];
        // Move to the pixel with the highest gradient value
        if (leftUp > left && leftUp > leftDown) {
            curr_x -= 1;
            curr_y -= 1;  // Move up-left
        } else if (leftDown > left && leftDown > leftUp) {
            curr_x -= 1;
            curr_y += 1;  // Move down-left
        } else {
            curr_x -= 1;  // Move straight-left
        }
    }
    drawEdgesFromAnchorKernel(curr_x, curr_y, d_gradient, d_direction, d_edge,
                              false, width, height);

    curr_x = x;
    curr_y = y;
    d_edge[y * width + x] = 0;
    while (validCuda(curr_x, curr_y, width, height) &&
           d_gradient[curr_y * width + curr_x] > 0 &&
           !d_edge[curr_y * width + curr_x] &&
           isHorizontalCuda(d_direction[curr_y * width + curr_x])) {
        d_edge[curr_y * width + curr_x] = 255;
        unsigned char rightUp = d_gradient[(curr_y - 1) * width + curr_x - 1];
        unsigned char right = d_gradient[curr_y * width + curr_x - 1];
        unsigned char rightDown = d_gradient[(curr_y + 1) * width + curr_x - 1];
        // Move to the pixel with the highest gradient value
        if (rightUp > right && rightUp > rightDown) {
            curr_x += 1;
            curr_y -= 1;  // Move up-right
        } else if (rightDown > right && rightDown > rightUp) {
            curr_x += 1;
            curr_y += 1;  // Move down-right
        } else {
            curr_x += 1;  // Move straight-right
        }
    }
    drawEdgesFromAnchorKernel(curr_x, curr_y, d_gradient, d_direction, d_edge,
                              false, width, height);
}

__device__ void drawVerticalEdgeFromAnchorKernel(int x, int y,
                                                 unsigned char *d_gradient,
                                                 float *d_direction,
                                                 unsigned char *d_edge,
                                                 int width, int height) {
    if (!validCuda(x, y, width, height)) return;

    int curr_x = x;
    int curr_y = y;
    d_edge[y * width + x] = 0;  // Assuming white edges on a black background

    // Trace upwards from the anchor point
    while (validCuda(curr_x, curr_y, width, height) &&
           d_gradient[curr_y * width + curr_x] > 0 &&
           !d_edge[curr_y * width + curr_x] &&
           !isHorizontalCuda(d_direction[curr_y * width + curr_x])) {
        d_edge[curr_y * width + curr_x] =
            255;  // Mark this pixel as part of an edge
        unsigned char upLeft = d_gradient[(curr_y - 1) * width + curr_x - 1];
        unsigned char up = d_gradient[(curr_y - 1) * width + curr_x];
        unsigned char upRight = d_gradient[(curr_y - 1) * width + curr_x + 1];

        // Move to the pixel with the highest gradient value above the current
        // pixel
        if (upLeft > up && upLeft > upRight) {
            curr_x -= 1;
            curr_y -= 1;  // Move top-left
        } else if (upRight > up && upRight > upLeft) {
            curr_x += 1;
            curr_y -= 1;  // Move top-right
        } else {
            curr_y -= 1;  // Move straight up
        }
    }
    drawEdgesFromAnchorKernel(curr_x, curr_y, d_gradient, d_direction, d_edge,
                              true, width, height);

    // Reset to anchor point
    curr_x = x;
    curr_y = y;
    d_edge[y * width + x] = 0;

    // Trace downwards from the anchor point
    while (validCuda(curr_x, curr_y, width, height) &&
           d_gradient[curr_y * width + curr_x] > 0 &&
           !d_edge[curr_y * width + curr_x] &&
           !isHorizontalCuda(d_direction[curr_y * width + curr_x])) {
        d_edge[curr_y * width + curr_x] =
            255;  // Mark this pixel as part of an edge
        unsigned char downLeft = d_gradient[(curr_y + 1) * width + curr_x - 1];
        unsigned char down = d_gradient[(curr_y + 1) * width + curr_x];
        unsigned char downRight = d_gradient[(curr_y + 1) * width + curr_x + 1];

        // Move to the pixel with the highest gradient value below the current
        // pixel
        if (downLeft > down && downLeft > downRight) {
            curr_x -= 1;
            curr_y += 1;  // Move bottom-left
        } else if (downRight > down && downRight > downLeft) {
            curr_x += 1;
            curr_y += 1;  // Move bottom-right
        } else {
            curr_y += 1;  // Move straight down
        }
    }
    drawEdgesFromAnchorKernel(curr_x, curr_y, d_gradient, d_direction, d_edge,
                              true, width, height);
}

__device__ void drawEdgesFromAnchorKernel(
    int x, int y, unsigned char *d_gradient, float *d_direction,
    unsigned char *d_edge, const bool horizontal, int width, int height) {
    // Check recursion base condition
    if (!validCuda(x, y, width, height) || d_gradient[y * width + x] <= 0 ||
        d_edge[y * width + x]) {
        return;
    }

    if (horizontal) {
        drawHorizontalEdgeFromAnchorKernel(x, y, d_gradient, d_direction,
                                           d_edge, width, height);
    } else {
        drawVerticalEdgeFromAnchorKernel(x, y, d_gradient, d_direction, d_edge,
                                         width, height);
    }
}

__global__ void drawEdgesFromAnchorsKernel(unsigned char *d_gradient,
                                           float *d_direction, bool *d_anchor,
                                           unsigned char *d_edge, int width,
                                           int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height && d_anchor[y * width + x]) {
        bool horizontal = isHorizontalCuda(d_direction[y * width + x]);
        drawEdgesFromAnchorKernel(x, y, d_gradient, d_direction, d_edge,
                                  horizontal, width, height);
    }
}

void drawEdgesFromAnchorsGPU(const CImg &gradient, const CImgFloat &direction,
                             const CImgBool &anchors, CImg &edge) {
    int width = gradient.width();
    int height = gradient.height();
    size_t numPixels = width * height;

    // Device memory pointers
    unsigned char *d_gradient;
    float *d_direction;
    bool *d_anchor;
    unsigned char *d_edge;

    // Allocate device memory
    hipMalloc(&d_gradient, numPixels * sizeof(unsigned char));
    hipMalloc(&d_direction, numPixels * sizeof(float));
    hipMalloc(&d_anchor, numPixels * sizeof(bool));
    hipMalloc(&d_edge, numPixels * sizeof(unsigned char));

    // Copy data to device
    hipMemcpy(d_gradient, gradient.data(), numPixels * sizeof(unsigned char),
               hipMemcpyHostToDevice);
    hipMemcpy(d_direction, direction.data(), numPixels * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(d_anchor, anchors.data(), numPixels * sizeof(bool),
               hipMemcpyHostToDevice);

    // Kernel launch parameters
    // TODO: extract true anchors for cuda threads
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                  (height + blockSize.y - 1) / blockSize.y);

    // Launch kernel
    drawEdgesFromAnchorsKernel<<<gridSize, blockSize>>>(
        d_gradient, d_direction, d_anchor, d_edge, width, height);

    // Copy results back to host
    hipMemcpy(edge.data(), d_edge, numPixels * sizeof(unsigned char),
               hipMemcpyDeviceToHost);
}