#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "edgedraw.h"

__constant__ int SOBEL_X[3][3] = {{-1, 0, 1}, {-2, 0, 2}, {-1, 0, 1}};
__constant__ int SOBEL_Y[3][3] = {{-1, -2, -1}, {0, 0, 0}, {1, 2, 1}};
__constant__ int SUPPRESS_THRESHOLD = GRADIENT_THRESH;

__global__ void colorToGrayKernel(unsigned char *image,
                                  unsigned char *grayImage, int width,
                                  int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int pixels = width * height;

    if (x < width && y < height) {
        int idx = y * width + x;
        unsigned char r = image[idx];
        unsigned char g = image[idx + pixels];
        unsigned char b = image[idx + 2 * pixels];
        unsigned char grayValue = 0.299f * r + 0.587f * g + 0.114f * b;
        grayImage[idx] = grayValue;
    }
}

__global__ void gradientCalculationKernel(unsigned char *grayImage,
                                          unsigned char *gradient,
                                          float *direction, int width,
                                          int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x > 0 && x < width - 1 && y > 0 && y < height - 1) {
        int gradientX = 0, gradientY = 0;

        // Apply the Sobel filter
        for (int i = -1; i <= 1; i++) {
            for (int j = -1; j <= 1; j++) {
                int pixel = grayImage[(y + j) * width + (x + i)];
                gradientX += SOBEL_X[i + 1][j + 1] * pixel;
                gradientY += SOBEL_Y[i + 1][j + 1] * pixel;
            }
        }

        int idx = y * width + x;
        gradient[idx] = sqrtf(gradientX * gradientX + gradientY * gradientY);
        direction[idx] = atan2f(gradientY, gradientX) * 180 / M_PI;
    }
}

void gradientInGrayGPU(CImg &image, CImg &gradient, CImgFloat &direction) {
    int width = image.width(), height = image.height();

    // Flatten the image data for CUDA
    unsigned char *d_image, *d_grayImage, *d_gradient;
    float *d_direction;
    size_t imageSize = width * height * 3 * sizeof(unsigned char);
    size_t grayImageSize = width * height * sizeof(unsigned char);
    size_t directionSize = width * height * sizeof(float);

    hipMalloc(&d_image, imageSize);
    hipMalloc(&d_grayImage, grayImageSize);
    hipMalloc(&d_gradient, grayImageSize);
    hipMalloc(&d_direction, directionSize);

    hipMemcpy(d_image, image.data(), imageSize, hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                  (height + blockSize.y - 1) / blockSize.y);

    colorToGrayKernel<<<gridSize, blockSize>>>(d_image, d_grayImage, width,
                                               height);
    gradientCalculationKernel<<<gridSize, blockSize>>>(
        d_grayImage, d_gradient, d_direction, width, height);

    // Copy results back to host
    hipMemcpy(gradient.data(), d_gradient, grayImageSize,
               hipMemcpyDeviceToHost);
    hipMemcpy(direction.data(), d_direction, directionSize,
               hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_image);
    hipFree(d_grayImage);
    hipFree(d_gradient);
    hipFree(d_direction);
}

__global__ void suppressWeakGradientsKernel(unsigned char *gradient, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        int idx = y * width + x;  // Index into the 1D array representation of the image
        if (gradient[idx] <= SUPPRESS_THRESHOLD) {
            gradient[idx] = 0;
        }
    }
}


void suppressWeakGradientsGPU(CImg &gradient) {
    int width = gradient.width(), height = gradient.height();
    size_t numPixels = width * height;
    unsigned char *d_gradient;

    // Allocate GPU memory
    hipMalloc(&d_gradient, numPixels * sizeof(unsigned char));

    // Copy data from host to device
    hipMemcpy(d_gradient, gradient.data(), numPixels * sizeof(unsigned char), hipMemcpyHostToDevice);

    // Define block and grid sizes
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, 
                  (height + blockSize.y - 1) / blockSize.y);

    // Launch the kernel
    suppressWeakGradientsKernel<<<gridSize, blockSize>>>(d_gradient, width, height);

    // Copy the modified data back to the host
    hipMemcpy(gradient.data(), d_gradient, numPixels * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_gradient);
}
