#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <math.h>

#include <algorithm>
#include <iostream>

#include "processing.h"

const int THREAD_PER_BLK = 256;
const double GAUSSIAN_BLUR_SIGMA = 1.5;
const int GAUSSIAN_BLUR_RADIUS = 5;

__global__ void gaussian_blur_kernel(const unsigned char* inputImage,
                                     unsigned char* outputImage, int width,
                                     int height, double* kernel,
                                     int kernelWidth, int channels) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = (row * width + col) * channels;

    if (col < width && row < height) {
        for (int c = 0; c < channels; ++c) {
            float sum = 0.0f;
            float totalWeight = 0.0f;

            for (int i = -GAUSSIAN_BLUR_RADIUS; i <= GAUSSIAN_BLUR_RADIUS;
                 ++i) {
                for (int j = -GAUSSIAN_BLUR_RADIUS; j <= GAUSSIAN_BLUR_RADIUS;
                     ++j) {
                    int newRow = min(max(row + i, 0), height - 1);
                    int newCol = min(max(col + j, 0), width - 1);
                    int newIdx = (newRow * width + newCol) * channels + c;

                    float weight =
                        kernel[(i + GAUSSIAN_BLUR_RADIUS) * kernelWidth +
                               (j + GAUSSIAN_BLUR_RADIUS)];
                    sum += double(inputImage[newIdx]) * weight;
                    totalWeight += weight;
                }
            }

            if (totalWeight == 0.0f) {
                totalWeight += 0.0000000001f;
            }

            outputImage[idx + c] =
                static_cast<unsigned char>(sum / totalWeight);
        }
    }
}

unsigned char* gaussianBlur(const unsigned char* inputImage, int width,
                            int height, int channels) {
    double kernelWidth = (2 * GAUSSIAN_BLUR_RADIUS) + 1;
    double kernelSize = kernelWidth * kernelWidth;
    double* kernel = (double*)malloc(kernelSize * sizeof(double));

    double sum = 0.0;
    // Populate every position in the kernel with the respective Gaussian
    // distribution value
    for (int x = -GAUSSIAN_BLUR_RADIUS; x <= GAUSSIAN_BLUR_RADIUS; x++) {
        for (int y = -GAUSSIAN_BLUR_RADIUS; y <= GAUSSIAN_BLUR_RADIUS; y++) {
            // Implement Gaussian function
            double expNumerator = -(x * x + y * y);
            double expDenominator =
                2.0 * GAUSSIAN_BLUR_SIGMA * GAUSSIAN_BLUR_SIGMA;
            double eExpression = exp(expNumerator / expDenominator);
            double kernelValue =
                eExpression /
                (2.0 * M_PI * GAUSSIAN_BLUR_SIGMA * GAUSSIAN_BLUR_SIGMA);
            size_t index = (x + GAUSSIAN_BLUR_RADIUS) * kernelWidth + y +
                           GAUSSIAN_BLUR_RADIUS;
            kernel[index] = kernelValue;
            sum += kernelValue;
        }
    }

    // Normalize the kernel
    for (int i = 0; i < kernelWidth; i++) {
        for (int j = 0; j < kernelWidth; j++) {
            size_t index = i * kernelWidth + j;
            kernel[index] /= sum;
        }
    }

    unsigned char* outputImage = (unsigned char*)malloc(
        sizeof(unsigned char) * width * height * channels);
    /*
    for (int i = 0; i < width; i++) {
        for (int j = 0; j < height; j++) {
            outputImage[i * width + j] = inputImage[i * width + j];
        }
    }
    */

    // Allocate device memory
    int imageDataSize = sizeof(unsigned char) * width * height * channels;
    unsigned char* inputImageDevice;
    unsigned char* outputImageDevice;
    double* kernelDevice;
    hipMalloc(&inputImageDevice, imageDataSize);
    hipMemcpy(inputImageDevice, inputImage, imageDataSize,
               hipMemcpyHostToDevice);
    hipMalloc(&outputImageDevice, imageDataSize);
    hipMalloc(&kernelDevice, sizeof(double) * kernelWidth * kernelWidth);
    hipMemcpy(kernelDevice, kernel, sizeof(double) * kernelWidth * kernelWidth,
               hipMemcpyHostToDevice);

    // CUDA memory allocation and kernel invocation
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                  (height + blockSize.y - 1) / blockSize.y);
    gaussian_blur_kernel<<<gridSize, blockSize>>>(
        inputImageDevice, outputImageDevice, width, height, kernelDevice,
        kernelWidth, channels);

    // Copy the result back to host
    hipMemcpy(outputImage, outputImageDevice, imageDataSize,
               hipMemcpyDeviceToHost);
    // free up memory
    // Free device memory
    hipFree(inputImageDevice);
    hipFree(outputImageDevice);
    hipFree(kernelDevice);

    return outputImage;
}
